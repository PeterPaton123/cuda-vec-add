#include <iostream>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <ctime>

__global__ void AddInts(int *a, int *b, int count) {
  // Get the thread id
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < count) {
    a[id] += b[id];
  }
}

int main() {
  std::srand(time(NULL));
  int count = 1000;
  
  // Array a and b stored on host, hence h_a
  int *h_a = new int[count];
  int *h_b = new int[count];
  int *h_c = new int[count];

  for (int i = 0; i < count; i++) {
    h_a[i] = std::rand() % 1000;
    h_b[i] = std::rand() % 1000;
  }
  
  for(int i = 0; i < 5; i++) {
    std::cout << h_a[i] << " " << h_b[i] << std::endl;
  }

  // Array a and b stored on device, hence d_a
  int *d_a, *d_b;

  if (hipMalloc(&d_a, sizeof(int) * count) != hipSuccess) {
    std::cout << "CUDA Malloc failed" << std::endl;
    return -1;  
  }
  
  if (hipMalloc(&d_b, sizeof(int) * count) != hipSuccess) {
    hipFree(d_a);    
    std::cout << "CUDA Malloc failed" << std::endl;
    return -1;  
  }

  if (hipMemcpy(d_a, h_a, sizeof(int) * count, hipMemcpyHostToDevice) != hipSuccess or hipMemcpy(d_b, h_b, sizeof(int) * count, hipMemcpyHostToDevice) != hipSuccess){
    std::cout << "Could not copy" << std::endl;
    hipFree(d_a);
    hipFree(d_b);
  }
  

  AddInts<<<count / 256 + 1, 256>>>(d_a, d_b, count);
  
  if (hipMemcpy(h_c, d_a, sizeof(int) * count, hipMemcpyDeviceToHost) != hipSuccess) {
    std::cout << "Copy back to host from device failed!" << std::endl;
    delete []h_a;
    delete []h_b; 
    hipFree(d_a);
    hipFree(d_b);
  }
  
  for(int i = 0; i < 10; i++) {
    std::cout << h_a[i] << " + " << h_b[i] << " = " << h_c[i] << std::endl;
  }
 
  return 0;
}
